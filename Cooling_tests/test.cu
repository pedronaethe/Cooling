#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

texture<float, cudaTextureType4D> textureQ;

__global__ void fetchQKernel(float* result, int sizeP1, int sizeP2, int sizeP3, int sizeP4)
{
    int p1 = threadIdx.x;
    int p2 = blockIdx.x;
    int p3 = blockIdx.y;
    int p4 = blockIdx.z;

    float q = tex4D(textureQ, p1, p2, p3, p4);
    result[p1 + p2 * sizeP1 + p3 * sizeP1 * sizeP2 + p4 * sizeP1 * sizeP2 * sizeP3] = q;
}

int main()
{
    // Set the sizes of the parameters
    int sizeP1 = 10;
    int sizeP2 = 20;
    int sizeP3 = 30;
    int sizeP4 = 40;

    // Allocate memory for the result on the host
    int resultSize = sizeP1 * sizeP2 * sizeP3 * sizeP4;
    float* resultHost = (float*)malloc(resultSize * sizeof(float));

    // Allocate memory for the result on the device
    float* resultDevice;
    hipMalloc((void**)&resultDevice, resultSize * sizeof(float));

    // Bind the texture to the device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipBindTextureToArray(textureQ, resultDevice, channelDesc);

    // Set texture parameters
    textureQ.normalized = false;
    textureQ.filterMode = hipFilterModePoint;
    textureQ.addressMode[0] = hipAddressModeClamp;
    textureQ.addressMode[1] = hipAddressModeClamp;
    textureQ.addressMode[2] = hipAddressModeClamp;
    textureQ.addressMode[3] = hipAddressModeClamp;

    // Launch the kernel
    dim3 blockSize(sizeP1, 1, 1);
    dim3 gridSize(sizeP2, sizeP3, sizeP4);
    fetchQKernel<<<gridSize, blockSize>>>(resultDevice, sizeP1, sizeP2, sizeP3, sizeP4);

    // Copy the result back to the host
    hipMemcpy(resultHost, resultDevice, resultSize * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipUnbindTexture(textureQ);
    hipFree(resultDevice);
    free(resultHost);

    return 0;
}
