
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

hipTextureObject_t coolTexObj;
hipArray *cuCoolArray = 0;

// Load the cooling_table into the CPU Memory.
void Load_Cooling_Tables(float *cooling_table)
{
    double *radius_arr;
    double *ne_arr;
    double *te_arr;
    double *bmag_arr;
    double *cool_arr;

    double radius;
    double ne;
    double te;
    double bmag;
    double cool;

    int i = 0;
    int nw = 32;
    int nx = 32; // Number of Te data.
    int ny = 32; // Number of ne data.
    int nz = 32; // Number of Bmag data.

    FILE *infile;

    // Allocate arrays for temperature, electronic density and radius data.
    radius_arr = (double *)malloc(nw * nx * ny * nz * sizeof(double));
    ne_arr = (double *)malloc(nw * nx * ny * nz * sizeof(double));
    te_arr = (double *)malloc(nw * nx * ny * nz * sizeof(double));
    cool_arr = (double *)malloc(nw * nx * ny *  nz * sizeof(double));
    bmag_arr = (double *)malloc(nw * nx * ny * nz * sizeof(double));

    // Reading the cooling table
    infile = fopen("cooling_table_new.txt", "r");

    if (infile == NULL)
    {
        printf("Unable to open cooling file.\n");
        exit(1);
    }

    fscanf(infile, "%*[^\n]\n"); // this command is to ignore the first line.
    while (fscanf(infile, "%lf, %lf, %lf, %lf, %lf", &radius, &bmag, &ne, &te, &cool) == 5)
    {
        radius_arr[i] = radius;
        ne_arr[i] = ne;
        te_arr[i] = te;
        bmag_arr[i] = bmag;
        cool_arr[i] = cool;

        i++;
    }

    fclose(infile);
    // copy data from cooling array into the table
    for (i = 0; i < nw * nx * ny * nz; i++)
    {
        cooling_table[i] = float(cool_arr[i]);
    }

    // Free arrays used to read in table data
    free(radius_arr);
    free(ne_arr);
    free(te_arr);    
    free(bmag_arr);
    free(cool_arr);
    return;
}

void CreateTexture(void)
{

    float *cooling_table; //Device Array with cooling floats
    // number of elements in each variable
    const int nw = 32; //r
    const int nx = 32; //te
    const int ny = 32; //ne
    const int nz = 32; //bmag
    cooling_table = (float *)malloc(nw * nx * ny * nz * sizeof(float));
    Load_Cooling_Tables(cooling_table); //Loading Cooling Values into pointer
    //cudaArray Descriptor
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    //cuda Array
    hipArray *cuCoolArray;
    //checkCudaErrors(cudaMalloc3DArray(&cuCoolArray, &channelDesc, make_cudaExtent(nx*sizeof(float),ny,nz), 0));
    hipMalloc3DArray(&cuCoolArray, &channelDesc, make_hipExtent(nx*ny,nz, nw), 0);
    hipMemcpy3DParms copyParams = {0};

    //Array creation
    copyParams.srcPtr   = make_hipPitchedPtr((void *) cooling_table, nx * ny * sizeof(float), nx * ny, nz);
    copyParams.dstArray = cuCoolArray;
    copyParams.extent   = make_hipExtent(nx * ny, nz, nw);
    copyParams.kind     = hipMemcpyHostToDevice;
    //checkCudaErrors(cudaMemcpy3D(&copyParams));
    hipMemcpy3D(&copyParams);
    //Array creation End

    hipResourceDesc    texRes;
    memset(&texRes, 0, sizeof(texRes));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array  = cuCoolArray;
    hipTextureDesc     texDescr;
    memset(&texDescr, 0, sizeof(texDescr));
    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;   // clamp
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.addressMode[2] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;
    //checkCudaErrors(cudaCreateTextureObject(&coolTexObj, &texRes, &texDescr, NULL));}
    hipCreateTextureObject(&coolTexObj, &texRes, &texDescr, NULL);
    return;
}
__global__ void cooling_function(hipTextureObject_t my_tex, float a0, float a1, float a2, float a3)
{
    float v0, v1, v2, v3, v4, lambda;

    //Values for testing;
    v0 = a0; //R parameter
    v1 = a1; //Bmag parameter
    v2 = a2; //ne parameter
    v3 = a3; //te parameter
    printf("Values you chose:\n");
    printf("Radius = %f, Bmag = %f, ne = %f, Te = %f\n", v0, v1, v2, v3);

    //For the non normalized version only.
    //The remapping formula goes (variable - initial_value) * (N - 1)/(max_value - init_value)
    // const int nx = 70; //Number of te used to generate table
    // const int ny = 70; //Number of ne used to generate table
    // const int nz = 70; //Number of r used to generate table
    //v1 = round((v1 - 6) * (nz - 1)/6);
    //v2 = round((v2 - 12) * (ny - 1)/8);
    //v3 = round((v3 - 6) * (nx - 1)/4);
    //printf("a = %f, b = %f, c = %f\n", v1, v2, v3);

    // For the normalized version only.
    const int nw = 32; //Number of R used to generate table
    const int nx = 32; //Number of te used to generate table
    const int ny = 32; //Number of ne used to generate table
    const int nz = 32; //Number of Bmag used to generate table
     v0 = (round((v0 - 6) * (nz - 1)/3) + 0.5)/nw; //radius
     v1 = (round((v1 - 0) * (nz - 1)/10) + 0.5)/nz; // Bmag
     v4 = ((round((v3 - 4) * (nx - 1)/11) + 0.5) + round((v2 - 12) * (ny - 1)/10) * nx)/(nx * ny); //Te + ne

    printf("Coordinates in texture grid:\n");
    printf("radius = %f, Bmag = %f, ne = %f, te = %f, ne+te = = %f\n", v0, v1, v2, v3, v4);

    //For the non normalized version only.
    //lambda = tex3D<float>(coolTexObj, v3 + 0.5f, v2 + 0.5f, v1 + 0.5f); 

    // //For the normalized version only.
    lambda = tex3D<float>(my_tex, v4, v1, v0); 
    printf("Cooling value = %lf\n", lambda);
    return;
}

int main()
{
    float read0, read1, read2, read3;
    float loop = 100;
    char str[1];
    CreateTexture();
    while (loop > 1)
    {
	    printf("radiusvalue:\n");
	    scanf("%f", &read0);
	    printf("Bmag value:\n");
	    scanf("%f", &read1);
	    printf("ne value:\n");
	    scanf("%f", &read2);
	    printf("Te value:\n");
	    scanf("%f", &read3);
	    cooling_function<<<1, 1>>>(coolTexObj, read0, read1, read2, read3);
        sleep(1);
	    printf("Do you want to read other values? y/n\n");
	    scanf("%s", str);
	    if (strcmp(str, "n") == 0)
	    {
	    	loop = 0;
	    }
	}
    hipDestroyTextureObject(coolTexObj);
    return 0;
}
//DEPRECATED Texture Reference in CUDA 11.0
/*
//Texture and cudaArray declaration.
 texture<float, 3, cudaReadModeElementType> coolTexObj;
cudaArray *cuCoolArray = 0;


// Load the cooling_table into the CPU Memory.
void Load_Cooling_Tables(float *cooling_table)
{
    double *ne_arr;
    double *te_arr;
    double *bmag_arr;
    double *cool_arr;

    double ne;
    double te;
    double bmag;
    double cool;

    int i = 0;
    int nx = 100; // Number of Te data.
    int ny = 100; // Number of ne data.
    int nz = 100; // Number of Bmag data.

    FILE *infile;

    // Allocate arrays for temperature, electronic density and radius data.
    ne_arr = (double *)malloc(nx * ny * nz * sizeof(double));
    te_arr = (double *)malloc(nx * ny * nz * sizeof(double));
    cool_arr = (double *)malloc(nx * ny *  nz * sizeof(double));
    bmag_arr = (double *)malloc(nx * ny * nz * sizeof(double));

    // Reading the cooling table
    infile = fopen("cooling_table_log_mag.txt", "r"); // this command is to ignore the first line.

    if (infile == NULL)
    {
        printf("Unable to open cooling file.\n");
        exit(1);
    }

    fscanf(infile, "%*[^\n]\n");
    while (fscanf(infile, "%lf, %lf, %lf, %lf", &bmag, &ne, &te, &cool) == 4)
    {
        ne_arr[i] = ne;
        te_arr[i] = te;
        bmag_arr[i] = bmag;
        cool_arr[i] = cool;

        i++;
    }

    fclose(infile);
    // copy data from cooling array into the table
    for (i = 0; i < nx * ny * nz; i++)
    {
        cooling_table[i] = float(cool_arr[i]);
    }

    // Free arrays used to read in table data
    free(ne_arr);
    free(te_arr);    
    free(bmag_arr);
    free(cool_arr);
}

 // \brief Load the Cloudy cooling tables into texture memory on the GPU. 
void Load_Cuda_Textures()
{

    float *cooling_table;

    // number of elements in each variable
    const int nx = 100; //te
    const int ny = 100; //ne
    const int nz = 100; //bmag


    // allocate host arrays to be copied to textures
    cooling_table = (float *)malloc(nx* ny * nz * sizeof(float));

    // Load cooling tables into the host arrays
    Load_Cooling_Tables(cooling_table);

    // Allocate CUDA arrays in device memory
    // The value of 64 in the CUDA channel must be checked, otherwise use 32 for float.
    cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc(32, 0, 0, 0, cudaChannelFormatKindFloat);
    cudaExtent volumeSize = make_cudaExtent(nx, ny, nz);
    cudaMalloc3DArray(&cuCoolArray, &channelDesc, volumeSize);

    // Copy to device memory the cooling and heating arrays
    // in host memory
    cudaMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_cudaPitchedPtr((void *)cooling_table, nx * sizeof(float), nx, ny); 
    copyParams.dstArray = cuCoolArray;
    copyParams.extent = volumeSize;
    copyParams.kind = cudaMemcpyHostToDevice;
    cudaMemcpy3D(&copyParams);

    // Specify texture reference parameters (same for both tables)
    coolTexObj.addressMode[0] = cudaAddressModeClamp; // out-of-bounds fetches return border values
    coolTexObj.addressMode[1] = cudaAddressModeClamp; // out-of-bounds fetches return border values
    coolTexObj.addressMode[2] = cudaAddressModeClamp; // out-of-bounds fetches return border values
    coolTexObj.filterMode = cudaFilterModeLinear;     // bi-linear interpolation
    coolTexObj.normalized = true;                     // Normalization of logarithm scale going from 0 to 1

    // Command to bind the array into the texture
    cudaBindTextureToArray(coolTexObj, cuCoolArray);
    // Free the memory associated with the cooling tables on the host
    free(cooling_table);
}

void Free_Cuda_Textures()
{
    // unbind the cuda textures
    cudaUnbindTexture(coolTexObj);
    // Free the device memory associated with the cuda arrays
    cudaFreeArray(cuCoolArray);
}

//Function used to interpolate the values of the cooling table.
__global__ void cooling_function(float a1, float a2, float a3)
{
    float v1, v2, v3, lambda;

    //Values for testing;
    v1 = a1; //Bmag parameter
    v2 = a2; //ne parameter
    v3 = a3; //te parameter
    printf("Values you chose:\n");
    printf("Bmag = %f, ne = %f, Te = %f\n", v1, v2, v3);

    //For the non normalized version only.
    //The remapping formula goes (variable - initial_value) * (N - 1)/(max_value - init_value)
    // const int nx = 70; //Number of te used to generate table
    // const int ny = 70; //Number of ne used to generate table
    // const int nz = 70; //Number of r used to generate table
    //v1 = round((v1 - 6) * (nz - 1)/6);
    //v2 = round((v2 - 12) * (ny - 1)/8);
    //v3 = round((v3 - 6) * (nx - 1)/4);
    //printf("a = %f, b = %f, c = %f\n", v1, v2, v3);

    // For the normalized version only.
    const int nx = 100; //Number of te used to generate table
    const int ny = 100; //Number of ne used to generate table
    const int nz = 100; //Number of Bmag used to generate table
     v1 = (round((v1 - 0.1) * (nz - 1)/9.99) + 0.5)/nz;
     v2 = (round((v2 - 12) * (ny - 1)/10) + 0.5 )/ny;
     v3 = (round((v3 - 4) * (nx - 1)/11) + 0.5 )/nx;

    printf("Coordinates in texture grid:\n");
    printf("Bmag = %f, ne = %f, Te = %f\n", v1, v2, v3);

    //For the non normalized version only.
    //lambda = tex3D<float>(coolTexObj, v3 + 0.5f, v2 + 0.5f, v1 + 0.5f); 

    // //For the normalized version only.
    lambda = tex3D<float>(coolTexObj, v3, v2, v1); 
    printf("Cooling value = %lf\n", lambda);
    return;
}

int main()
{
    float read1, read2, read3;
    float loop = 100;
    char str[1];
    Load_Cuda_Textures();
    while (loop > 1)
    {
	    printf("Bmag value:\n");
	    scanf("%f", &read1);
	    printf("ne value:\n");
	    scanf("%f", &read2);
	    printf("Te value:\n");
	    scanf("%f", &read3);
	    cooling_function<<<1, 1>>>(read1, read2, read3);
        sleep(1);
	    printf("Do you want to read other values? y/n\n");
	    scanf("%s", str);
	    if (strcmp(str, "n") == 0)
	    {
	    	loop = 0;
	    }
	}
    Free_Cuda_Textures();

    return 0;
}*/
