
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


#define SIZEOF_H 101 /*Size of H's in your cooling table*/
#define SIZEOF_B 101 /*Size of B's in your cooling table*/
#define SIZEOF_TE 101 /*Size of Te's in your cooling table*/
#define SIZEOF_NE 101/*Size of Ne's in your cooling table*/
#define N_RESOLUTION 12600 /*This is for resolution_test and is the number of cells in your simulation*/
#define DT 7.336005915070878e-07 /*This is an approximation of the timestep for coulomb test*/

#define THOMSON_CGS (6.652e-25) /*Thomson's cross section in CGS*/
#define BOLTZ_CGS (1.3806504e-16) /*Boltzmann constant in CGS*/
#define TABLE_SIZE (SIZEOF_H * SIZEOF_B * SIZEOF_TE * SIZEOF_NE) /*Total size of the table*/
#define SIZEOF_TEST 130 /*Quad root of number of calculations for GLOBAL_MEMORY_TEST*/

#define SINGLE_TEST (0) /*Single value test*/
#define RESOLUTION_TEST (1) /*Compare analytical values with values from the table*/
#define COMPARISON_MARCEL (0) /*Compare plot A.1 of Marcel et al. 2018: A unified accretion-ejection paradigm for black hole X-ray binaries*/
#define GLOBAL_MEMORY_TEST (0) /*Test texture memory vs global memory efficiency*/
#define INDEX(i, j, k, l) (l + SIZEOF_TE * (k + SIZEOF_NE * (j + SIZEOF_B * i))) /*4D indexing*/

/*Declaration of both texture objects*/
hipTextureObject_t coolTexObj;
hipArray *cuCoolArray = 0;

hipTextureObject_t coulombTexObj;
hipArray *cuCoulombArray = 0;

/*This function loads the cooling values from the binary file*/
void Load_Cooling_Tables(float *cooling_table)
{
    fprintf(stderr, "Loading Table...\n");

    int i = 0;
    int nw = SIZEOF_H; //Number of H data
    int nx = SIZEOF_TE; // Number of Te data.
    int ny = SIZEOF_NE; // Number of ne data.
    int nz = SIZEOF_B;  // Number of Bmag data.

    FILE *infile;
    double value;

    // Reading the cooling table
    infile = fopen("cooling_table_05.bin", "rb");

    if (infile == NULL)
    {
        fprintf(stderr, "Unable to open cooling file.\n");
        exit(1);
    }
    fprintf(stderr, "Reading Data...\n");

    // Opening the binary file and reading the data, while also transferring it to the pointer cooling_table
    for (i = 0; i < nw * nx * ny * nz; i++)
    {
        fread(&value, sizeof(double), 1, infile);
        cooling_table[i] = float(value);
    }

    fprintf(stderr, "Finished transfering .binary data to memory!\n");
    fclose(infile);

    printf("Table Loaded!\n");

    return;
}

/*This function will transfer the values from the table to the texture object*/
void CreateTexture(void)
{
    float *cooling_table; //Pointer that will hold the cooling values from Load_cooling_table function
    const int nw = SIZEOF_H;  // Number of H data
    const int nx = SIZEOF_TE; // Number of Te data
    const int ny = SIZEOF_NE; // Number of ne data
    const int nz = SIZEOF_B;  // Number of Bmag data
    cooling_table = (float *)malloc(nw * nx * ny * nz * sizeof(float)); //Allocating memory for cooling_table pointer

    Load_Cooling_Tables(cooling_table); // Loading Cooling Values into pointer
    
    // cudaArray Descriptor
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    // cuda Array
    hipArray *cuCoolArray;

    //Creating 3D array in device memory  
    hipMalloc3DArray(&cuCoolArray, &channelDesc, make_hipExtent(nx * ny, nz, nw), 0);
    hipMemcpy3DParms copyParams = {0};

    // Copying cooling values from host memory to device array.
    copyParams.srcPtr = make_hipPitchedPtr((void *)cooling_table, nx * ny* sizeof(float), nx * ny, nz);
    copyParams.dstArray = cuCoolArray;
    copyParams.extent = make_hipExtent(nx * ny, nz, nw);
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);
    // Array creation End

    //Defining parameters for the texture object
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(texRes));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = cuCoolArray;
    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(texDescr));
    texDescr.normalizedCoords = false; //Whether to use normalized coordinates or not, this will impact the indexing
    texDescr.filterMode = hipFilterModeLinear; // Whether to use nearest-neighbor approximation or trilinear interpolation
    texDescr.addressMode[0] = hipAddressModeClamp; // Out of boundary conditions in dimension 1
    texDescr.addressMode[1] = hipAddressModeClamp; // Out of boundary conditions in dimension 2
    texDescr.addressMode[2] = hipAddressModeClamp; // Out of boundary conditions in dimension 3
    texDescr.readMode = hipReadModeElementType; // Type of values stored in texture object

    hipCreateTextureObject(&coolTexObj, &texRes, &texDescr, NULL); //Creating the texture object with the channel and parameters described above
    printf("Texture Created!\n");
    return;
}

/*This function creates an array of linear equally spaced values*/
__device__ void linspace(float start, float end, int numPoints, float *result)
{
    float stepSize = (end - start) / (float)(numPoints - 1);
    for (int i = 0; i < numPoints; i++)
    {
        result[i] = start + i * stepSize;
    }
}

__global__ void cooling_function(hipTextureObject_t my_tex, float a0, float a1, float a2, float a3)
{
    float v0, v1, v4;
    double lambda;

    const int nw = SIZEOF_H;  // Number of H used to generate table
    const int nx = SIZEOF_TE; // Number of te used to generate table
    const int ny = SIZEOF_NE; // Number of ne used to generate table
    const int nz = SIZEOF_B;  // Number of Bmag used to generate table

    // We need this to do our manual interpolation. listofa1 will hold values of magfield from the table and listofa2 will hold values of ne.
    float listofa1[SIZEOF_B]; 
    float listofa2[SIZEOF_NE];
    float a1_index, a2_index;

    // Generate the values used in the table by both parameters B and ne
    linspace(0, 10, SIZEOF_B, listofa1);
    linspace(2, 25, SIZEOF_NE, listofa2);

    // Calculate both dimensions that are not flattened
    v0 = (floor(((a0 - 3.) > 0 ? a0 - 3. : 0) * (nw - 1.) / 5.) + 0.5);
    v4 = (floor(((a3 - 2.) > 0 ? a3 - 2. : 0) * (nx - 1.) / 13.) + 0.5);

    // printf("lambda = %lf",lambda);
    //In order not to mess up with our interpolation, if the values for the magnetic field and density extrapolates, we'll set them to their max value.
    if (a1 > 10)
    {
        a1 = 10;
    }
    else if (a2 > 25)
    {
        a2 = 25;
    }

    // These will give us the indexing of B and ne from the table, we gotta see if they are integers or not.
    a1_index = (((a1 - 0.) > 0 ? a1 : 0) * (nz - 1.) / 10.);
    a2_index = (((a2 - 2.) > 0 ? a2 - 2. : 0) * (ny - 1.) / 23.);


    /*Now we manually interpolate between B and ne region. If B value is equal to a table value, we don't need to interpolate it and its index will be an integer.
    However, if it's not, the index will not be a an integer and we need to take the fractional part and interpolate, we adress the four possible scenarios:
    indexB and indexNe are both integers, indexB is an integer but indexNe is not, indexNe is an integer but indexB is not and none of them are integers.*/
    if (a1_index == (int)a1_index && a2_index == (int)a2_index) //condition for both of them being integers
    {
        printf("Entrance 1 \n");
        v1 = ((((a2 - 2.) > 0 ? a2 - 2. : 0) * (ny - 1.) / 23.) + ((a1 - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
        printf("v0 = %lf, v1 = %lf, v4 = %lf\n", v0, v1, v4);
        lambda = tex3D<float>(my_tex, v4, v1, v0);
    }
    else if (a1_index != (int)a1_index && a2_index != (int)a2_index)//condition for none of them being integers
    {
        printf("Entrance 2 \n");
        float alpha, beta, v1_ij, v1_i1j, v1_ij1, v1_i1j1;
        alpha = a1_index - floor(a1_index);
        beta = a2_index - floor(a2_index);

        v1_ij = (floor(((a2 - 2.) > 0 ? a2 - 2. : 0) * (ny - 1.) / 23.) + floor((a1 - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
        v1_i1j = ((floor(((a2 - 2.) > 0 ? a2 - 2. : 0) * (ny - 1.) / 23.) + 1) + floor((a1 - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
        v1_ij1 = ((floor(((a2 - 2.) > 0 ? a2 - 2. : 0) * (ny - 1.) / 23.)) + (floor((a1 - 0.) * (nz - 1.) / 10.) + 1) * (ny) + 0.5);
        v1_i1j1 = ((floor(((a2 - 2.) > 0 ? a2 - 2. : 0) * (ny - 1.) / 23.) + 1) + (floor((a1 - 0.) * (nz - 1.) / 10.) + 1) * (ny) + 0.5);
        printf("B_before = %lf, B_after = %lf, ne_before = %lf, ne_after = %lf\n", floor(a1_index), floor(a1_index) + 1, floor(a2_index), floor(a2_index) + 1);
        printf("alpha = %lf, beta = %lf, texij = %lf, texi1j = %lf, texij1 = %lf, texi1j1 = %lf \n", alpha, beta, tex3D<float>(my_tex, v4, v1_ij, v0),
               tex3D<float>(my_tex, v4, v1_i1j, v0), tex3D<float>(my_tex, v4, v1_ij1, v0), tex3D<float>(my_tex, v4, v1_i1j1, v0));

        lambda = (1 - alpha) * (1 - beta) * tex3D<float>(my_tex, v4, v1_ij, v0) + alpha * (1 - beta) * tex3D<float>(my_tex, v4, v1_i1j, v0) +
                 (1 - alpha) * beta * tex3D<float>(my_tex, v4, v1_ij1, v0) + alpha * beta * tex3D<float>(my_tex, v4, v1_i1j1, v0);
    }
    else if (a1_index != (int)a1_index) //Condition for indexB not integer and indexNe being an integer
    {
        printf("Entrance 3 \n");
        float alpha, v1_i, v1_i1;
        printf("a1_index = %lf\n", a1_index);
        alpha = a1_index - floor(a1_index);
        v1_i = ((((a2 - 2.) > 0 ? a2 - 2. : 0) * (ny - 1.) / 23.) + floor((a1 - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
        v1_i1 = ((((a2 - 2.) > 0 ? a2 - 2. : 0) * (ny - 1.) / 23.) + (floor((a1 - 0.) * (nz - 1.) / 10.) + 1) * (ny) + 0.5);
        printf("v1_i = %lf, v1_i1 = %lf \n", v1_i, v1_i1);
        printf("alpha = %lf, tex before = %lf, tex after = %lf \n", alpha, tex3D<float>(my_tex, v4, v1_i, v0), tex3D<float>(my_tex, v4, v1_i1, v0));
        lambda = (1 - alpha) * tex3D<float>(my_tex, v4, v1_i, v0) + alpha * tex3D<float>(my_tex, v4, v1_i1, v0);
    }
    else //Condition for indexNe not integer and indexB being an integer
    {
        printf("Entrance 4 \n");
        float alpha, v1_i, v1_i1;
        alpha = a2_index - floor(a2_index);
        v1_i = (floor(((a2 - 2.) > 0 ? a2 - 2. : 0) * (ny - 1.) / 23.) + ((a1 - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
        v1_i1 = ((floor(((a2 - 2.) > 0 ? a2 - 2. : 0) * (ny - 1.) / 23.) + 1) + ((a1 - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
        printf("v1_i = %lf, v1_i1 = %lf \n", v1_i, v1_i1);
        printf("alpha = %lf, tex before = %lf, tex after = %lf \n", alpha, tex3D<float>(my_tex, v4, v1_i, v0), tex3D<float>(my_tex, v4, v1_i1, v0));
        lambda = (1 - alpha) * tex3D<float>(my_tex, v4, v1_i, v0) + alpha * tex3D<float>(my_tex, v4, v1_i1, v0);
    }

    printf("Coordinates in texture grid:\n");
    printf("Cooling value = %lf\n", lambda);
    return;
}

__global__ void cooling_function_new(hipTextureObject_t my_tex, float a0, float a1, float a2, float a3)
{
    float v0, v1, v4;
    double lambda;
    float a2_index, a3_index;
    // double t_break = 9.472016;
    // double t_ubreak = 9.540000;
    // double t_lbreak = 9.410000;
    double t_break = 9.472016;
    double t_ubreak = 9.71875; //para 101
    double t_lbreak = 9.3125; //para 101
    float alpha, beta, v4_ij, v4_i1j, v4_ij1, v4_i1j1, v4_i, v4_i1;
    float v4_ihalfj, v4_ihalfj1, v4_im1j1, v4_im1j, v4_iM1j1, v4_iM1j, v4_im2j1, v4_im2j, v4_iM2j1, v4_iM2j, frac_break, alpha_lower, alpha_upper;

    const int nw = SIZEOF_H;  // Number of H used to generate table
    const int nx = SIZEOF_TE; // Number of te used to generate table
    const int ny = SIZEOF_NE; // Number of ne used to generate table
    const int nz = SIZEOF_B;  // Number of Bmag used to generate table

        // Calculate both dimensions that are not flattened
        v0 = ((((a0 - 3.) > 0 ? a0 - 3. : 0) * (nw - 1.) / 5.) + 0.5);
        v1 = (((a1 - 0.) > 0 ? a1 : 0) * (nz - 1.) / 10. + 0.5);


        // Select maximum values separetly
        if (a2 > 25)
        {
            a2 = 25;
        }
        else if (a3 > 15)
        {
            a3 = 15;
        }

        // These will give us the indexing of B and ne from the table, we gotta see if they are integers or not.
        a3_index = (((a3 - 2.) > 0 ? a3 - 2. : 0) * (nx - 1.) / 13.);
        a2_index = (((a2 - 2.) > 0 ? a2 - 2. : 0) * (ny - 1.) / 23.);
        if (a3_index == (int)a3_index && a2_index == (int)a2_index)
        {//working
            printf("Entrada 1, ambos inteiros\n");
            v4 = ((a3_index) + a2_index * (nx) + 0.5);
            lambda = tex3D<float>(my_tex, v4, v1, v0);
        }
        else if (a3_index != (int)a3_index && a2_index != (int)a2_index)
        {   
            printf("Entrada 2, Nenhum inteiro\n");
            beta = a2_index - floor(a2_index);
            alpha = a3_index - floor(a3_index);
            if (a3 < t_break && a3 > t_lbreak){
                printf("temperatura entre o break e o limite inferior\n"); //working
                frac_break = t_break - t_lbreak;
                alpha_lower = (a3 - t_lbreak)/frac_break;
                v4_ij = (floor(a3_index) + floor(a2_index) * (nx) + 0.5);
                v4_ij1 = ((floor(a3_index)) + (floor(a2_index) + 1) * (nx) + 0.5);


                v4_im2j =((floor(a3_index) - 2) + (floor(a2_index))  * (nx) + 0.5);
                v4_im1j = ((floor(a3_index) - 1) + (floor(a2_index)) * (nx) + 0.5);
                v4_ihalfj = 3 * tex3D<float>(my_tex, v4_ij, v1, v0) - 3 * tex3D<float>(my_tex, v4_im1j, v1, v0) + tex3D<float>(my_tex, v4_im2j, v1, v0);

                v4_im2j1 =((floor(a3_index) - 2) + (floor(a2_index) + 1)  * (nx) + 0.5);
                v4_im1j1 = ((floor(a3_index) - 1) + (floor(a2_index) + 1) * (nx) + 0.5);
                v4_ihalfj1 = 3 * tex3D<float>(my_tex, v4_ij1, v1, v0) - 3 * tex3D<float>(my_tex, v4_im1j1, v1, v0) + tex3D<float>(my_tex, v4_im2j1, v1, v0);
                printf("alpha_lower = %lf, beta = %lf v4_ihalfj = %lf, v4_ihalfj1 = %lf \n", alpha_lower, beta, v4_ihalfj, v4_ihalfj1);
                printf("T(i-2, j) = %lf, T(i-1, j) = %lf, T(i,j) = %lf\n", tex3D<float>(my_tex, v4_im2j, v1, v0), tex3D<float>(my_tex, v4_im1j, v1, v0), tex3D<float>(my_tex, v4_ij, v1, v0));
                printf("T(i-2, j+1) = %lf, T(i-1, j+1) = %lf, T(i, j+1) = %lf\n", tex3D<float>(my_tex, v4_im2j1, v1, v0), tex3D<float>(my_tex, v4_im1j1, v1, v0), tex3D<float>(my_tex, v4_ij1, v1, v0));

                lambda = (1 - alpha_lower) * (1 - beta) * tex3D<float>(my_tex, v4_ij, v1, v0) + alpha_lower * (1 - beta) * v4_ihalfj +
                         (1 - alpha_lower) * beta * tex3D<float>(my_tex, v4_ij1, v1, v0) + alpha_lower * beta * v4_ihalfj1;                

            }else if(a3 >t_break && a3 < t_ubreak){//working
                printf("temperatura entre o break e o limite superior\n");
                frac_break = t_ubreak - t_break;
                alpha_upper = (a3 - t_break)/(frac_break);
                v4_ij = ((floor(a3_index) + 1) + floor(a2_index) * (nx) + 0.5);
                v4_ij1 = ((floor(a3_index) + 1) + (floor(a2_index) + 1) * (nx) + 0.5);


                v4_iM2j =((floor(a3_index) + 3) + (floor(a2_index))  * (nx) + 0.5);
                v4_iM1j = ((floor(a3_index) + 2) + (floor(a2_index)) * (nx) + 0.5);
                v4_ihalfj = 3 * tex3D<float>(my_tex, v4_ij, v1, v0) - 3 * tex3D<float>(my_tex, v4_iM1j, v1, v0) + tex3D<float>(my_tex, v4_iM2j, v1, v0);

                v4_iM2j1 =((floor(a3_index) + 3) + (floor(a2_index) + 1)  * (nx) + 0.5);
                v4_iM1j1 = ((floor(a3_index) + 2) + (floor(a2_index) + 1) * (nx) + 0.5);
                v4_ihalfj1 = 3 * tex3D<float>(my_tex, v4_ij1, v1, v0) - 3 * tex3D<float>(my_tex, v4_iM1j1, v1, v0) + tex3D<float>(my_tex, v4_iM2j1, v1, v0);
                printf("alpha_upper = %lf, beta = %lf v4_ihalfj = %lf, v4_ihalfj1 = %lf \n", alpha_upper, beta, v4_ihalfj, v4_ihalfj1);
                printf("T(i+2, j) = %lf, T(i+1, j) = %lf, T(i,j) = %lf\n", tex3D<float>(my_tex, v4_iM2j, v1, v0), tex3D<float>(my_tex, v4_iM1j, v1, v0), tex3D<float>(my_tex, v4_ij, v1, v0));
                printf("T(i+2, j+1) = %lf, T(i+1, j+1) = %lf, T(i, j+1) = %lf\n", tex3D<float>(my_tex, v4_iM2j1, v1, v0), tex3D<float>(my_tex, v4_iM1j1, v1, v0), tex3D<float>(my_tex, v4_ij1, v1, v0));
                lambda = (1 - alpha_upper) * (1 - beta) * v4_ihalfj + alpha_upper * (1 - beta) * tex3D<float>(my_tex, v4_ij, v1, v0) +
                         (1 - alpha_upper) * beta * v4_ihalfj1 + alpha_upper * beta * tex3D<float>(my_tex, v4_ij1, v1, v0);  
            }else{//working
                printf("temperatura longe do break\n");
                v4_ij = (floor(a3_index) + floor(a2_index) * (nx) + 0.5);
                v4_i1j = ((floor(a3_index) + 1) + floor(a2_index) * (nx) + 0.5);
                v4_ij1 = ((floor(a3_index)) + (floor(a2_index) + 1) * (nx) + 0.5);
                v4_i1j1 = ((floor(a3_index) + 1) + (floor(a2_index) + 1) * (nx) + 0.5);
                printf("v4_ij = %lf, v4_i1j = %lf, v4_ij1 = %lf, v4_i1j1 = %lf, alpha = %lf, beta = %lf \n", v4_ij, v4_i1j, v4_ij1, v4_i1j1, alpha, beta);
                printf("ij value = %lf, i1j value = %lf, ij1 value = %lf, i1j1 value = %lf \n",tex3D<float>(my_tex, v4_ij, v1, v0), tex3D<float>(my_tex, v4_i1j, v1, v0), tex3D<float>(my_tex, v4_ij1, v1, v0), tex3D<float>(my_tex, v4_i1j1, v1, v0));
                lambda = (1 - alpha) * (1 - beta) * tex3D<float>(my_tex, v4_ij, v1, v0) + alpha * (1 - beta) * tex3D<float>(my_tex, v4_i1j, v1, v0) +
                        (1 - alpha) * beta * tex3D<float>(my_tex, v4_ij1, v1, v0) + alpha * beta * tex3D<float>(my_tex, v4_i1j1, v1, v0);
            }
        }
        else if (a2_index != (int)a2_index) //Condition for indexne not integer and indexte being an integer
        { //working
            printf("Entrada 3, ne não é inteiro \n");
            alpha = a2_index - floor(a2_index);
            v4_i = ((a3_index) + floor(a2_index) * (nx) + 0.5);
            v4_i1 = ((a3_index) + (floor(a2_index) + 1) * (nx) + 0.5);
            lambda = (1 - alpha) * tex3D<float>(my_tex, v4_i, v1, v0) + alpha * tex3D<float>(my_tex, v4_i1, v1, v0);
        }
        else //Condition for indexte not integer and indexne being an integer
        { //working
            printf("Entrada 4, Te não é inteiro \n");
            if (a3 < t_break && a3 > t_lbreak){
                printf("temperatura entre o break e o limite inferior\n"); 
                frac_break = t_break - t_lbreak;
                alpha_lower = (a3 - t_lbreak)/frac_break;
                v4_ij = (floor(a3_index) + (a2_index) * (nx) + 0.5);
                v4_im2j =((floor(a3_index) - 2) + (floor(a2_index))  * (nx) + 0.5);
                v4_im1j = ((floor(a3_index) - 1) + (floor(a2_index)) * (nx) + 0.5);
                v4_ihalfj = 3 * tex3D<float>(my_tex, v4_ij, v1, v0) - 3 * tex3D<float>(my_tex, v4_im1j, v1, v0) + tex3D<float>(my_tex, v4_im2j, v1, v0);
                printf("T(i-2) = %lf, T(i-1) = %lf, T(i) = %lf\n", tex3D<float>(my_tex, v4_im2j, v1, v0), tex3D<float>(my_tex, v4_im1j, v1, v0), tex3D<float>(my_tex, v4_ij, v1, v0));
                printf("alpha = %lf, frac_break = %lf, alpha_upper = %lf, v4_ij = %lf, v4_iM2j = %lf, v4_iM1j = %lf, v4_ihalfj = %lf \n", alpha, frac_break, alpha_lower, v4_ij, v4_im2j, v4_im1j, v4_ihalfj);

                lambda = (1 - alpha_lower) * tex3D<float>(my_tex, v4_ij, v1, v0) + alpha_lower * v4_ihalfj;
            }else if(a3 >t_break && a3 < t_ubreak){ //working
                printf("temperatura entre o break e o limite superior\n"); //quanto menor o alpha, maior a importancia do T[i];
                frac_break = t_ubreak - t_break;
                alpha_upper = (a3 - t_break)/(frac_break);
                v4_ij = (floor(a3_index + 1) + floor(a2_index) * (nx) + 0.5);
                v4_iM2j =((floor(a3_index) + 3) + (floor(a2_index))  * (nx) + 0.5);
                v4_iM1j = ((floor(a3_index) + 2) + (floor(a2_index)) * (nx) + 0.5);
                v4_ihalfj = 3 * tex3D<float>(my_tex, v4_ij, v1, v0) - 3 * tex3D<float>(my_tex, v4_iM1j, v1, v0) + tex3D<float>(my_tex, v4_iM2j, v1, v0);
                printf("T(i+2) = %lf, T(i+1) = %lf, T(i) = %lf\n", tex3D<float>(my_tex, v4_iM2j, v1, v0), tex3D<float>(my_tex, v4_iM1j, v1, v0), tex3D<float>(my_tex, v4_ij, v1, v0));
                printf("alpha = %lf, frac_break = %lf, alpha_upper = %lf, v4_ij = %lf, v4_iM2j = %lf, v4_iM1j = %lf, v4_ihalfj = %lf \n", alpha, frac_break, alpha_upper, v4_ij, v4_iM2j, v4_iM1j, v4_ihalfj);
                lambda = (1 - alpha_upper) * v4_ihalfj + alpha_upper * tex3D<float>(my_tex, v4_i1j, v1, v0);

            }else{ //working
                printf("temperatura longe do break\n");
                alpha = a3_index - floor(a3_index);
                v4_i = (floor(a3_index) + (a2_index) * (nx) + 0.5);
                v4_i1 = ((floor(a3_index) + 1) + (a2_index) * (nx) + 0.5);
                printf("v4_i = %lf, v4_i1 = %lf, alpha = %lf \n", v4_i, v4_i1, alpha);
                lambda = (1 - alpha) * tex3D<float>(my_tex, v4_i, v1, v0) + alpha * tex3D<float>(my_tex, v4_i1, v1, v0);
            }
        }

    printf("Coordinates in texture grid:\n");
    printf("Cooling value = %lf\n", lambda);
    return;
}

__global__ void cooling_function_marcel(hipTextureObject_t my_tex, float a0, double *a1, double *a2, double *value)
{
    float v0, v1, v4;
    double ne_test, B_test, mu = 0.1;
    const int nw = SIZEOF_H;  // Number of H used to generate table
    const int nx = SIZEOF_TE; // Number of te used to generate table
    const int ny = SIZEOF_NE; // Number of ne used to generate table
    const int nz = SIZEOF_B;  // Number of Bmag used to generate table
    //  v0 = (round((v0 - 3) * (nz - 1)/5) + 0.5)/nw; //scale_height
    //  v1 = (round((v1 - 0) * (nz - 1)/10) + 0.5)/nz; // Bmag
    //  v4 = ((round((v3 -2) * (nx - 1)/13) + 0.5) + round((v2 - 10) * (ny - 1)/15) * nx)/(nx * ny); //Te + ne
    for (int i = 0; i < 20; i++)
    {
        ne_test = a1[i] / (a0 * THOMSON_CGS);
        for (int k = 0; k < 20; k++)
        {
            B_test = sqrt(2 * mu * BOLTZ_CGS * ne_test * a2[k]);
            // v0 = (round((log10(a0) - 3.) * (nw - 1.)/5.) + 0.5)/nw;
            // v1 = (round((log10(ne_test) - 10.) * (ny - 1.)/15.) + 0.5 + round((log10(B_test) - 0.) * (nz - 1.)/10.) * ny)/(nz * ny);
            // v4 = (round((log10(a2[k]) - 2.) * (nx - 1.)/13.) + 0.5)/nx;
            v0 = (round((log10(a0) - 5.) * (nw - 1.) / 3.) + 0.5) / nw;
            v1 = (round((log10(ne_test) - 8.) * (ny - 1.) / 15.) + 0.5 + round((log10(B_test) - 0.) * (nz - 1.) / 10.) * ny) / (nz * ny);
            v4 = (round((log10(a2[k]) - 2.) * (nx - 1.) / 13.) + 0.5) / nx;
            value[20 * i + k] = tex3D<float>(my_tex, v4, v1, v0);
        }
    }
    return;
}

__global__ void cooling_function_test_new(hipTextureObject_t my_tex, double *a0, double *a1, double *a2, double *a3, double *value)
{
    double v0, v1, v4;
    double lambda;
    int i;
    float a2_index, a3_index;

    double t_break = 9.472016;
    double t_ubreak = 9.540000;
    double t_lbreak = 9.410000;

    // double t_break = 9.472016;
    // double t_ubreak = 9.71875; //para 101
    // double t_lbreak = 9.3125; //para 101
    
    float alpha, beta, v4_ij, v4_i1j, v4_ij1, v4_i1j1, v4_i, v4_i1;
    float v4_ihalfj, v4_ihalfj1, v4_im1j1, v4_im1j, v4_iM1j1, v4_iM1j, v4_im2j1, v4_im2j, v4_iM2j1, v4_iM2j, frac_break, alpha_lower, alpha_upper;

    // For the normalized version only.
    const int nw = SIZEOF_H;  // Number of H used to generate table
    const int nx = SIZEOF_TE; // Number of te used to generate table
    const int ny = SIZEOF_NE; // Number of ne used to generate table
    const int nz = SIZEOF_B;  // Number of Bmag used to generate table
    for (i = 0; i < N_RESOLUTION; i++){
    
        // Calculate both dimensions that are not flattened
        v0 = ((((a0[i] - 3.) > 0 ? a0[i] - 3. : 0) * (nw - 1.) / 5.) + 0.5);
        v1 = (((a1[i] - 0.) > 0 ? a1[i] : 0) * (nz - 1.) / 10. + 0.5);


        // Select maximum values separetly
        if (a2[i] > 25)
        {
            a2[i] = 25;
        }
        else if (a3[i] > 15)
        {
            a3[i] = 15;
        }

        // These will give us the indexing of B and ne from the table, we gotta see if they are integers or not.
        a3_index = (((a3[i] - 2.) > 0 ? a3[i] - 2. : 0) * (nx - 1.) / 13.);
        a2_index = (((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.);
  
        if (a3_index == (int)a3_index && a2_index == (int)a2_index)
        {
            v4 = ((a3_index) + a2_index * (nx) + 0.5);
            lambda = tex3D<float>(my_tex, v4, v1, v0);
        }
        else if (a3_index != (int)a3_index && a2_index != (int)a2_index)
        {   
            beta = a2_index - floor(a2_index);
            alpha = a3_index - floor(a3_index);
            if (a3[i] < t_break && a3[i] > t_lbreak){
                frac_break = t_break - t_lbreak;
                alpha_lower = (a3[i] - t_lbreak)/frac_break;
                v4_ij = (floor(a3_index) + floor(a2_index) * (nx) + 0.5);
                v4_ij1 = ((floor(a3_index)) + (floor(a2_index) + 1) * (nx) + 0.5);


                v4_im2j =((floor(a3_index) - 2) + (floor(a2_index))  * (nx) + 0.5);
                v4_im1j = ((floor(a3_index) - 1) + (floor(a2_index)) * (nx) + 0.5);
                v4_ihalfj = 3 * tex3D<float>(my_tex, v4_ij, v1, v0) - 3 * tex3D<float>(my_tex, v4_im1j, v1, v0) + tex3D<float>(my_tex, v4_im2j, v1, v0);

                v4_im2j1 =((floor(a3_index) - 2) + (floor(a2_index) + 1)  * (nx) + 0.5);
                v4_im1j1 = ((floor(a3_index) - 1) + (floor(a2_index) + 1) * (nx) + 0.5);
                v4_ihalfj1 = 3 * tex3D<float>(my_tex, v4_ij1, v1, v0) - 3 * tex3D<float>(my_tex, v4_im1j1, v1, v0) + tex3D<float>(my_tex, v4_im2j1, v1, v0);

                lambda = (1 - alpha_lower) * (1 - beta) * tex3D<float>(my_tex, v4_ij, v1, v0) + alpha_lower * (1 - beta) * v4_ihalfj +
                         (1 - alpha_lower) * beta * tex3D<float>(my_tex, v4_ij1, v1, v0) + alpha_lower * beta * v4_ihalfj1;                
          

            }else if(a3[i] >t_break && a3[i] < t_ubreak){//
                frac_break = t_ubreak - t_break;
                alpha_upper = (a3[i] - t_break)/(frac_break);
                v4_ij = ((floor(a3_index) + 1) + floor(a2_index) * (nx) + 0.5);
                v4_ij1 = ((floor(a3_index) + 1) + (floor(a2_index) + 1) * (nx) + 0.5);


                v4_iM2j =((floor(a3_index) + 3) + (floor(a2_index))  * (nx) + 0.5);
                v4_iM1j = ((floor(a3_index) + 2) + (floor(a2_index)) * (nx) + 0.5);
                v4_ihalfj = 3 * tex3D<float>(my_tex, v4_ij, v1, v0) - 3 * tex3D<float>(my_tex, v4_iM1j, v1, v0) + tex3D<float>(my_tex, v4_iM2j, v1, v0);

                v4_iM2j1 =((floor(a3_index) + 3) + (floor(a2_index) + 1)  * (nx) + 0.5);
                v4_iM1j1 = ((floor(a3_index) + 2) + (floor(a2_index) + 1) * (nx) + 0.5);
                v4_ihalfj1 = 3 * tex3D<float>(my_tex, v4_ij1, v1, v0) - 3 * tex3D<float>(my_tex, v4_iM1j1, v1, v0) + tex3D<float>(my_tex, v4_iM2j1, v1, v0);
                lambda = (1 - alpha_upper) * (1 - beta) * v4_ihalfj + alpha_upper * (1 - beta) * tex3D<float>(my_tex, v4_ij, v1, v0) +
                         (1 - alpha_upper) * beta * v4_ihalfj1 + alpha_upper * beta * tex3D<float>(my_tex, v4_ij1, v1, v0);  
            }else{//
                v4_ij = (floor(a3_index) + floor(a2_index) * (nx) + 0.5);
                v4_i1j = ((floor(a3_index) + 1) + floor(a2_index) * (nx) + 0.5);
                v4_ij1 = ((floor(a3_index)) + (floor(a2_index) + 1) * (nx) + 0.5);
                v4_i1j1 = ((floor(a3_index) + 1) + (floor(a2_index) + 1) * (nx) + 0.5);
                lambda = (1 - alpha) * (1 - beta) * tex3D<float>(my_tex, v4_ij, v1, v0) + alpha * (1 - beta) * tex3D<float>(my_tex, v4_i1j, v1, v0) +
                        (1 - alpha) * beta * tex3D<float>(my_tex, v4_ij1, v1, v0) + alpha * beta * tex3D<float>(my_tex, v4_i1j1, v1, v0);
            }
        }
        else if (a2_index != (int)a2_index) //Condition for indexne not integer and indexte being an integer
        {//
            alpha = a2_index - floor(a2_index);
            v4_i = ((a3_index) + floor(a2_index) * (nx) + 0.5);
            v4_i1 = ((a3_index) + (floor(a2_index) + 1) * (nx) + 0.5);
            lambda = (1 - alpha) * tex3D<float>(my_tex, v4_i, v1, v0) + alpha * tex3D<float>(my_tex, v4_i1, v1, v0);
        }
        else //Condition for indexte not integer and indexne being an integer
        {
            alpha = a2_index - floor(a2_index);
            if (a3[i] < t_break && a3[i] > t_lbreak){//
                frac_break = t_break - t_lbreak;
                alpha_lower = (a3[i] - t_lbreak)/frac_break;
                v4_ij = (floor(a3_index) + (a2_index) * (nx) + 0.5);
                v4_im2j =((floor(a3_index) - 2) + (floor(a2_index))  * (nx) + 0.5);
                v4_im1j = ((floor(a3_index) - 1) + (floor(a2_index)) * (nx) + 0.5);
                v4_ihalfj = 3 * tex3D<float>(my_tex, v4_ij, v1, v0) - 3 * tex3D<float>(my_tex, v4_im1j, v1, v0) + tex3D<float>(my_tex, v4_im2j, v1, v0);

                lambda = (1 - alpha_lower) * tex3D<float>(my_tex, v4_ij, v1, v0) + alpha_lower * v4_ihalfj;
                lambda = (1 - alpha) * tex3D<float>(my_tex, v4_ij, v1, v0) + alpha * v4_ihalfj;
            }else if(a3[i] >t_break && a3[i] < t_ubreak){//
                alpha_upper = (a3[i] - t_break)/(t_ubreak - t_break);
                v4_ij = (floor(a3_index + 1) + floor(a2_index) * (nx) + 0.5);
                v4_iM2j =((floor(a3_index) + 3) + (floor(a2_index))  * (nx) + 0.5);
                v4_iM1j = ((floor(a3_index) + 2) + (floor(a2_index)) * (nx) + 0.5);
                v4_ihalfj = 3 * tex3D<float>(my_tex, v4_ij, v1, v0) - 3 * tex3D<float>(my_tex, v4_iM1j, v1, v0) + tex3D<float>(my_tex, v4_iM2j, v1, v0);
                lambda = (1 - alpha_upper) * v4_ihalfj + alpha_upper * tex3D<float>(my_tex, v4_i1j, v1, v0);

            }else{//
                alpha = a3_index - floor(a3_index);
                v4_i = (floor(a3_index) + (a2_index) * (nx) + 0.5);
                v4_i1 = ((floor(a3_index) + 1) + (a2_index) * (nx) + 0.5);
                lambda = (1 - alpha) * tex3D<float>(my_tex, v4_i, v1, v0) + alpha * tex3D<float>(my_tex, v4_i1, v1, v0);
            }
        }
        value[i] = lambda;
    }
    
    return;
}

__global__ void cooling_function_test(hipTextureObject_t my_tex, double *a0, double *a1, double *a2, double *a3, double *value)
{
    double v0, v1, v4;
    double lambda;
    int i;

    // For the normalized version only.
    const int nw = SIZEOF_H;  // Number of H used to generate table
    const int nx = SIZEOF_TE; // Number of te used to generate table
    const int ny = SIZEOF_NE; // Number of ne used to generate table
    const int nz = SIZEOF_B;  // Number of Bmag used to generate table
    for (i = 0; i < N_RESOLUTION; i++)
    {


        // Because we are going to interpolate manually, we need to define the lists that will hold the value for the parameters (same as table)
        float listofa1[SIZEOF_B];
        float listofa2[SIZEOF_NE];
        float a1_index, a2_index;

        // Generate the values used in the table by both parameters B and ne
        linspace(0, 10, SIZEOF_B, listofa1);
        linspace(2, 25, SIZEOF_NE, listofa2);

        // Calculate both dimensions that are not flattened
        v0 = (floor(((a0[i] - 3.) > 0 ? a0[i] - 3. : 0) * (nw - 1.) / 5.) + 0.5);

        //Interpolação da temperatura devido a não continuidade da função síncrotron, a quebra acontece em
        double t_break = 7.7730466;
        double t_ubreak = 7.85;
        double t_lbreak = 7.72;


         if (a3[i] < t_break && a3[i] > t_lbreak){
            double v4_indexim2 =(floor(((a3[i] - 2.) > 0 ? a3[i] - 2. : 0) * (nx - 1.) / 13.) - 1 + 0.5); //= (((a3[i] - 2.) > 0 ? a3[i] - 2. : 0) * (nx - 1.) / 13.);
            double v4_indexim1 = (floor(((a3[i] - 2.) > 0 ? a3[i] - 2. : 0) * (nx - 1.) / 13.) + 0.5);
            double 
            v4 = (floor(((a3[i] - 2.) > 0 ? a3[i] - 2. : 0) * (nx - 1.) / 13.) + 0.5);
        }else if (a3[i] >t_break && a3[i] < t_ubreak){
            v4 =  (floor(((a3[i] - 2.) > 0 ? a3[i] - 2. : 0) * (nx - 1.) / 13.) + 1 + 0.5);
        }else{
            v4 = (floor(((a3[i] - 2.) > 0 ? a3[i] - 2. : 0) * (nx - 1.) / 13.) + 0.5);
        }

        // if (a3[i] < 7.7730466 && a3[i] > 7.72){
        //     v4 = (floor(((a3[i] - 2.) > 0 ? a3[i] - 2. : 0) * (nx - 1.) / 13.) + 0.5);
        // }else if (a3[i] >7.7730466 && a3[i] < 7.85){
        //     v4 =  (floor(((a3[i] - 2.) > 0 ? a3[i] - 2. : 0) * (nx - 1.) / 13.) + 1 + 0.5);
        // }else{
        //     v4 = (floor(((a3[i] - 2.) > 0 ? a3[i] - 2. : 0) * (nx - 1.) / 13.) + 0.5);
        // }
        //v4 = (floor(((a3[i] - 2.) > 0 ? a3[i] - 2. : 0) * (nx - 1.) / 13.) + 0.5);
        // Select maximum values separetly
        if (a1[i] > 10)
        {
            a1[i] = 10;
        }
        else if (a2[i] > 25)
        {
            a2[i] = 25;
        }

        //v1 = (floor(((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.) + floor((a1[i] - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);

        // These will give us the indexing of B and ne from the table, we gotta see if they are integers or not.
        a1_index = (((a1[i] - 0.) > 0 ? a1[i] : 0) * (nz - 1.) / 10.);
        a2_index = (((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.);

        if (a1_index == (int)a1_index && a2_index == (int)a2_index)
        {
            v1 = ((((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.) + ((a1[i] - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
            lambda = tex3D<float>(my_tex, v4, v1, v0);
        }
        else if (a1_index != (int)a1_index && a2_index != (int)a2_index)
        {
            float alpha, beta, v1_ij, v1_i1j, v1_ij1, v1_i1j1;
            beta = a1_index - floor(a1_index);
            alpha = a2_index - floor(a2_index);

            v1_ij = (floor(((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.) + floor((a1[i] - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
            v1_i1j = ((floor(((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.) + 1) + floor((a1[i] - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
            v1_ij1 = ((floor(((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.)) + (floor((a1[i] - 0.) * (nz - 1.) / 10.) + 1) * (ny) + 0.5);
            v1_i1j1 = ((floor(((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.) + 1) + (floor((a1[i] - 0.) * (nz - 1.) / 10.) + 1) * (ny) + 0.5);

            lambda = (1 - alpha) * (1 - beta) * tex3D<float>(my_tex, v4, v1_ij, v0) + alpha * (1 - beta) * tex3D<float>(my_tex, v4, v1_i1j, v0) +
                     (1 - alpha) * beta * tex3D<float>(my_tex, v4, v1_ij1, v0) + alpha * beta * tex3D<float>(my_tex, v4, v1_i1j1, v0);
        }
        else if (a1_index != (int)a1_index) //Condition for indexB not integer and indexNe being an integer
        {
            float alpha, v1_i, v1_i1;
            alpha = a1_index - floor(a1_index);
            v1_i = ((((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.) + floor((a1[i] - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
            v1_i1 = ((((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.) + (floor((a1[i] - 0.) * (nz - 1.) / 10.) + 1) * (ny) + 0.5);
            lambda = (1 - alpha) * tex3D<float>(my_tex, v4, v1_i, v0) + alpha * tex3D<float>(my_tex, v4, v1_i1, v0);
        }
        else //Condition for indexNe not integer and indexB being an integer
        {
            float alpha, v1_i, v1_i1;
            alpha = a2_index - floor(a2_index);
            v1_i = (floor(((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.) + ((a1[i] - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
            v1_i1 = ((floor(((a2[i] - 2.) > 0 ? a2[i] - 2. : 0) * (ny - 1.) / 23.) + 1) + ((a1[i] - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
            lambda = (1 - alpha) * tex3D<float>(my_tex, v4, v1_i, v0) + alpha * tex3D<float>(my_tex, v4, v1_i1, v0);
        }
        value[i] = lambda;
    }
    return;
}

/*This function utilizes binary search to search for the closest index to the table for GLOBAL_MEMORY_TEST, used to find global_memory indexing*/
__device__ int binarySearchClosest(double *tablevalue, int size, double target)
{
    int left = 0;
    int right = size - 1;

    while (left <= right)
    {
        int mid = left + (right - left) / 2;

        // Check if the target is present at the middle
        if (tablevalue[mid] == target)
            return mid;

        // If target greater, ignore left half
        if (tablevalue[mid] < target)
            left = mid + 1;

        // If target is smaller, ignore right half
        else
            right = mid - 1;
    }

    // Find the index of the closest element
    if (abs(tablevalue[left] - target) < abs(tablevalue[right] - target))
        return left;
    else
        return right;
}

__global__ void global_memory_reading(double *parameterH, double *parameterB, double *parameterNe, double *parameterTe, double *H_list, double *B_list, double *ne_list, double *Te_list, double *cooling, double *value)
{
    for (int i = 0; i < SIZEOF_TEST; i++)
    {
        int indexH = binarySearchClosest(H_list, SIZEOF_H, parameterH[i]);
        for (int j = 0; j < SIZEOF_TEST; j++)
        {
            int indexB = binarySearchClosest(B_list, SIZEOF_B, parameterB[j]);
            for (int k = 0; k < SIZEOF_TEST; k++)
            {
                int indexNe = binarySearchClosest(ne_list, SIZEOF_NE, parameterNe[k]);
                for (int l = 0; l < SIZEOF_TEST; l++)
                {
                    int indexTe = binarySearchClosest(Te_list, SIZEOF_TE, parameterTe[l]);
                    value[INDEX(indexH, indexB, indexNe, indexTe)] = cooling[INDEX(indexH, indexB, indexNe, indexTe)];
                }
            }
        }
    }
}

void logspace(double start, double end, int num, double *result)
{
    double log_start = log10(start);                 // Initial value
    double log_end = log10(end);                     // End value
    double step = (log_end - log_start) / (num - 1); // number of steps
    int i;
    for (i = 0; i < num; ++i)
    {
        result[i] = log_start + i * step;
    }
}

__global__ void cooling_function_comparison_global(hipTextureObject_t my_tex, double *a0, double *a1, double *a2, double *a3, double *value)
{
    double v0, v1, v4;
    double lambda;
    // For the normalized version only.
    const int nw = SIZEOF_H;  // Number of H used to generate table
    const int nx = SIZEOF_TE; // Number of te used to generate table
    const int ny = SIZEOF_NE; // Number of ne used to generate table
    const int nz = SIZEOF_B;  // Number of Bmag used to generate table

    for (int i = 0; i < SIZEOF_TEST; i++)
    {
        for (int j = 0; j < SIZEOF_TEST; j++)
        {
            for (int k = 0; k < SIZEOF_TEST; k++)
            {
                for (int l = 0; l < SIZEOF_TEST; l++)
                {
                    v0 = (floor(((a0[i] - 5.) > 0 ? a0[i] - 5. : 0) * (nw - 1.) / 3.) + 0.5);
                    v1 = (floor(((a2[k] - 2.) > 0 ? a2[k] - 2. : 0) * (ny - 1.) / 23.) + floor((a1[j] - 0.) * (nz - 1.) / 10.) * (ny) + 0.5);
                    v4 = (floor(((a3[l] - 2.) > 0 ? a3[l] - 2. : 0) * (nx - 1.) / 13.) + 0.5);
                    lambda = tex3D<float>(my_tex, v4, v1, v0);
                    value[INDEX(i, j, k, l)] = lambda;
                }
            }
        }
    }
    return;
}
int main()
{
#if (SINGLE_TEST)
    float read0, read1, read2, read3;
    float loop = 100;
    float *value;

    char str[1];
    CreateTexture();
    while (loop > 1)
    {
        fprintf(stderr,"scale_height value:\n");
        scanf("%f", &read0);
        fprintf(stderr,"Bmag value:\n");
        scanf("%f", &read1);
        fprintf(stderr,"ne value:\n");
        scanf("%f", &read2);
        fprintf(stderr,"Te value:\n");
        scanf("%f", &read3);
        cooling_function_new<<<1, 1>>>(coolTexObj, read0, read1, read2, read3);
        cudaDeviceSynchronize();
        fprintf(stderr,"Do you want to read other values? y/n\n");
        scanf("%s", str);
        if (strcmp(str, "n") == 0)
        {
            loop = 0;
        }
    }
    cudaDestroyTextureObject(coolTexObj);
#elif (RESOLUTION_TEST)
    double *H_test, *B_test, *ne_test, *Te_test, *cool_test;
    H_test = (double *)malloc(N_RESOLUTION * sizeof(double));
    B_test = (double *)malloc(N_RESOLUTION * sizeof(double));
    ne_test = (double *)malloc(N_RESOLUTION * sizeof(double));
    Te_test = (double *)malloc(N_RESOLUTION * sizeof(double));
    cool_test = (double *)malloc(N_RESOLUTION * sizeof(double));
    int i;

    // Allocating memory in device memory.
    double *d_H_test;
    hipMalloc(&d_H_test, N_RESOLUTION * sizeof(double));
    double *d_B_test;
    hipMalloc(&d_B_test, N_RESOLUTION * sizeof(double));
    double *d_ne_test;
    hipMalloc(&d_ne_test, N_RESOLUTION * sizeof(double));
    double *d_Te_test;
    hipMalloc(&d_Te_test, N_RESOLUTION * sizeof(double));
    double *d_cool_test;
    hipMalloc(&d_cool_test, N_RESOLUTION * sizeof(double));


    fprintf(stderr, "Initializing resolution test reading\n");
    char filename[] = "New_005.txt";
    FILE *file_result;
    file_result = fopen(filename, "w");
    FILE *file_height_test;
    file_height_test = fopen("scaleheight_sim.txt", "r");
    FILE *file_e_density_test;
    file_e_density_test = fopen("electronic_density_sim.txt", "r");
    FILE *file_temperature_test;
    file_temperature_test = fopen("electronic_temperature_sim.txt", "r");
    FILE *file_mag_field_test;
    file_mag_field_test = fopen("magnetic_field_sim.txt", "r");
    // FILE *file_ucov_test;
    // file_ucov_test = fopen("ucov_sim.txt", "r");
    // FILE *file_ug_test;
    // file_ug_test = fopen("ug_sim.txt", "r");
    CreateTexture();
    for (i = 0; fscanf(file_height_test, "%lf", H_test + i) == 1; i++)
    {
        // Do nothing inside the loop body, everything is done in the for loop header
    }
    for (i = 0; fscanf(file_mag_field_test, "%lf", B_test + i) == 1; i++)
    {
        // Do nothing inside the loop body, everything is done in the for loop header
    }
    for (i = 0; fscanf(file_e_density_test, "%lf", ne_test + i) == 1; i++)
    {
        // Do nothing inside the loop body, everything is done in the for loop header
    }
    for (i = 0; fscanf(file_temperature_test, "%lf", Te_test + i) == 1; i++)
    {
        // Do nothing inside the loop body, everything is done in the for loop header
    }

    hipMemcpy(d_H_test, H_test, N_RESOLUTION * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B_test, B_test, N_RESOLUTION * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_ne_test, ne_test, N_RESOLUTION * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Te_test, Te_test, N_RESOLUTION * sizeof(double), hipMemcpyHostToDevice);
    fprintf(stderr, "Reading and getting values from texture memory...\n");

    cooling_function_test_new<<<1, 1>>>(coolTexObj, d_H_test, d_B_test, d_ne_test, d_Te_test, d_cool_test);
    hipMemcpy(cool_test, d_cool_test, N_RESOLUTION * sizeof(double), hipMemcpyDeviceToHost);

    for (i = 0; i < N_RESOLUTION; i++)
    {
        fprintf(file_result, "%.8e\n", cool_test[i]);
    }

    free(H_test);
    free(B_test);
    free(ne_test);
    free(Te_test);
    free(cool_test);
    hipDestroyTextureObject(coolTexObj);

    hipFree(d_H_test);
    hipFree(d_B_test);
    hipFree(d_ne_test);
    hipFree(d_Te_test);
    hipFree(d_cool_test);

    fclose(file_height_test);
    fclose(file_e_density_test);
    fclose(file_temperature_test);
    fclose(file_mag_field_test);
    fprintf(stderr, "Resolution test sucessfull, table generated: %s\n", filename);
#elif (COMPARISON_MARCEL)
    CreateTexture();
    double *te_test, H = 0.1 * 1.483366675977058e6 * 30, *tau_test, *result;
    double *tau_test_d, *te_test_d, *result_d;
    cudaMalloc(&tau_test_d, 20 * sizeof(double));
    cudaMalloc(&te_test_d, 20 * sizeof(double));
    cudaMalloc(&result_d, 400 * sizeof(double));

    tau_test = (double *)malloc(20 * sizeof(double)); // Allocate memory for tau_test on the host
    te_test = (double *)malloc(20 * sizeof(double));
    result = (double *)malloc(400 * sizeof(double));

    double tau_start = 1.e-6, tau_end = 5.e2;
    double te_start = 5.e4, te_end = 2.e11;
    FILE *file_result;
    file_result = fopen("marcel_comp.txt", "w");
    logspace(tau_start, tau_end, 20, tau_test);
    logspace(te_start, te_end, 20, te_test);

    cudaMemcpy(tau_test_d, tau_test, 20 * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(te_test_d, te_test, 20 * sizeof(double), cudaMemcpyHostToDevice);

    cooling_function_marcel<<<1, 1>>>(coolTexObj, H, tau_test_d, te_test_d, result_d);
    cudaMemcpy(result, result_d, 400 * sizeof(double), cudaMemcpyDeviceToHost);

    for (int i = 0; i < 400; i++)
    {
        fprintf(file_result, "%.8e,", pow(10., result[i]));
    }
    fclose(file_result);
    // free(te_test);
    // free(tau_test);
#elif (GLOBAL_MEMORY_TEST)
    double *H_test, *B_test, *ne_test, *Te_test, *cool_test;
    double *H_random, *B_random, *ne_random, *Te_random;
    double *results = 0;
    clock_t start_time, end_time;
    double duration;
    cudaError_t error;

    H_test = (double *)malloc(SIZEOF_H * sizeof(double));
    B_test = (double *)malloc(SIZEOF_B * sizeof(double));
    ne_test = (double *)malloc(SIZEOF_NE * sizeof(double));
    Te_test = (double *)malloc(SIZEOF_TE * sizeof(double));
    cool_test = (double *)malloc(TABLE_SIZE * sizeof(double));

    H_random = (double *)malloc(SIZEOF_TEST * sizeof(double));
    B_random = (double *)malloc(SIZEOF_TEST * sizeof(double));
    ne_random = (double *)malloc(SIZEOF_TEST * sizeof(double));
    Te_random = (double *)malloc(SIZEOF_TEST * sizeof(double));

    int i;

    // Allocating memory in device memory.
    double *d_H_test;
    cudaMalloc(&d_H_test, SIZEOF_H * sizeof(double));
    double *d_B_test;
    cudaMalloc(&d_B_test, SIZEOF_B * sizeof(double));
    double *d_ne_test;
    cudaMalloc(&d_ne_test, SIZEOF_NE * sizeof(double));
    double *d_Te_test;
    cudaMalloc(&d_Te_test, SIZEOF_TE * sizeof(double));
    double *d_cool_test;
    cudaMalloc(&d_cool_test, TABLE_SIZE * sizeof(double));

    double *d_H_random;
    cudaMalloc(&d_H_random, SIZEOF_TEST * sizeof(double));
    double *d_B_random;
    cudaMalloc(&d_B_random, SIZEOF_TEST * sizeof(double));
    double *d_Ne_random;
    cudaMalloc(&d_Ne_random, SIZEOF_TEST * sizeof(double));
    double *d_Te_random;
    cudaMalloc(&d_Te_random, SIZEOF_TEST * sizeof(double));
    double *d_results;
    cudaMalloc(&d_results, SIZEOF_TEST * sizeof(double));
    fprintf(stderr,"Initializing GLOBAL MEMORY testing\n");
    FILE *file_height_test;
    file_height_test = fopen("scale_height.txt", "r");
    FILE *file_e_density_test;
    file_e_density_test = fopen("ne.txt", "r");
    FILE *file_temperature_test;
    file_temperature_test = fopen("te.txt", "r");
    FILE *file_mag_field_test;
    file_mag_field_test = fopen("mag.txt", "r");
    FILE *file_cooling_test;
    file_cooling_test = fopen("cooling_table.bin", "rb");
    for (i = 0; fscanf(file_height_test, "%lf", H_test + i) == 1; i++)
    {
        // Do nothing inside the loop body, everything is done in the for loop header
    }
    for (i = 0; fscanf(file_mag_field_test, "%lf", B_test + i) == 1; i++)
    {
        // Do nothing inside the loop body, everything is done in the for loop header
    }
    for (i = 0; fscanf(file_e_density_test, "%lf", ne_test + i) == 1; i++)
    {
        // Do nothing inside the loop body, everything is done in the for loop header
    }
    for (i = 0; fscanf(file_temperature_test, "%lf", Te_test + i) == 1; i++)
    {
        // Do nothing inside the loop body, everything is done in the for loop header
    }
    for (i = 0; i < TABLE_SIZE; i++)
    {
        // fprintf(stderr,"i = %d \n", i);
        fread(&cool_test[i], sizeof(double), 1, file_cooling_test);
        // fprintf(stderr,"cool_test[%d] = %lf\n", i, cool_test[i]);
    }
    logspace(1e5, 1e8, SIZEOF_TEST, H_random);
    logspace(1e0, 1e10, SIZEOF_TEST, B_random);
    logspace(1e2, 1e15, SIZEOF_TEST, Te_random);
    logspace(1e2, 1e25, SIZEOF_TEST, ne_random);
    fprintf(stderr,"Transfering data from Host to Device... \n");
    cudaMemcpy(d_H_test, H_test, SIZEOF_H * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(d_B_test, B_test, SIZEOF_B * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(d_ne_test, ne_test, SIZEOF_NE * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(d_Te_test, Te_test, SIZEOF_TE * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(d_cool_test, cool_test, TABLE_SIZE * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(d_H_random, H_random, SIZEOF_TEST * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(d_B_random, B_random, SIZEOF_TEST * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(d_Ne_random, ne_random, SIZEOF_TEST * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(d_Te_random, Te_random, SIZEOF_TEST * sizeof(double), cudaMemcpyHostToDevice);
    CreateTexture();

    fprintf(stderr,"Starting table lookup...\n");
    start_time = clock();
    global_memory_reading<<<1, 1>>>(d_H_random, d_B_random, d_Ne_random, d_Te_random, d_H_test, d_B_test, d_ne_test, d_Te_test, d_cool_test, d_results);
    cudaDeviceSynchronize();
    end_time = clock();
    duration = (double)(end_time - start_time) / CLOCKS_PER_SEC;
    fprintf(stderr,"Number of values analyzed = %d, duration global: %.6f seconds\n", SIZEOF_TEST, duration);

    start_time = clock();
    cooling_function_comparison_global<<<1, 1>>>(coolTexObj, d_H_random, d_B_random, d_Ne_random, d_Te_random, d_results);
    cudaDeviceSynchronize();
    end_time = clock();
    duration = (double)(end_time - start_time) / CLOCKS_PER_SEC;
    fprintf(stderr,"Number of values analyzed = %d, duration texture: %.6f seconds\n", SIZEOF_TEST, duration);

#endif
    return 0;
}
